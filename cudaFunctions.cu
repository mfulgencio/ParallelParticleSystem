#include "hip/hip_runtime.h"
/* cudaFunctions.cu
 *
 * Contains the code for all the cuda functions used
 *
 */
 
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include "ParticleSystem.h"
#include "Util/SVector3.h"
#include "Util/SSphere.h"

#define THREADS_PER_BLOCK 32

// Variables needed for moveParticle(): ParticleSystem.particles[], ParticleSystem.speed,
   // particle index, float "time" variable, GRAVITY constant
// Variables needed for resetParticle(): ParticleSystem.particles[], particle index,
   // SVector3 translation, rbFloat() & rFloat(), SPEED & YSPEED constants
   
typedef struct {
   Particle particles[MAX_PARTICLES];
   int numParticles;
   float speed;
   float random;
   SVector3 Translation;
} CudaParticleSystem;


__device__ float rFloat(hiprandState *randStates)  {
   int id = (blockIdx.x * blockDim.x) + threadIdx.x;
   hiprandState localState = randStates[id];
   float randNum = (float)hiprand_uniform(&localState);
   randStates[id] = localState;
   
   return randNum;
}


__device__ float rbFloat(hiprandState *randStates) {
   return rFloat(randStates) * 2 - 1;
}


__device__ void moveParticle(Particle particle, float speed, float time) {
   particle.velocity.Y -= GRAVITY * time;

   particle.sphere.center.X += particle.velocity.X * time * speed; 
   particle.sphere.center.Y += particle.velocity.Y * time * speed; 
   particle.sphere.center.Z += particle.velocity.Z * time * speed; 
}


__device__ void resetParticle(hiprandState *randStates, Particle particle, SVector3 Translation, float random, float time) {
   particle.sphere.center.X = Translation.X + rbFloat(randStates) * random;
   particle.sphere.center.Y = Translation.Y + rbFloat(randStates) * random;
   particle.sphere.center.Z = Translation.Z + rbFloat(randStates) * random;
   
   particle.velocity.X = rbFloat(randStates) * SPEED;
   particle.velocity.Y = -rFloat(randStates) * YSPEED;
   particle.velocity.Z = rbFloat(randStates) * SPEED;
}


__global__ void update(hiprandState *randStates, CudaParticleSystem *cpsys, float time) {
   int index = (blockIdx.x * blockDim.x) + threadIdx.x;
   Particle curParticle = cpsys->particles[index];
   
   moveParticle(curParticle, cpsys->speed, time);
   
   if(curParticle.sphere.center.Y < -2) {
      hiprand_init(1234, index, 0, &randStates[index]);
      resetParticle(randStates, curParticle, cpsys->Translation, cpsys->random, time);
   }
}


extern "C" void cudaUpdate(ParticleSystem *psys, float time) {
   int num_blocks = 0;
   hiprandState *randStates;

   CudaParticleSystem *cpsys_device;
   
   hipMalloc((void **)&cpsys_device, sizeof(CudaParticleSystem));
   hipMalloc((void **)&randStates, MAX_PARTICLES * sizeof(hiprandState));
   
   hipMemcpy(cpsys_device->particles, psys->particles, sizeof(Particle) * MAX_PARTICLES, hipMemcpyHostToDevice);
   hipMemcpy(&cpsys_device->numParticles, &psys->numParticles, sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(&cpsys_device->speed, &psys->speed, sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(&cpsys_device->random, &psys->random, sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(&cpsys_device->Translation, &psys->Translation, sizeof(SVector3), hipMemcpyHostToDevice);
   
   num_blocks = ceil(MAX_PARTICLES / THREADS_PER_BLOCK);
   
   update<<<THREADS_PER_BLOCK, num_blocks>>>(randStates, cpsys_device, time);
   
   
   hipMemcpy(psys->particles, cpsys_device->particles, sizeof(Particle) * MAX_PARTICLES, hipMemcpyDeviceToHost);
}




