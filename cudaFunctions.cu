#include "hip/hip_runtime.h"
/* cudaFunctions.cu
 *
 * Contains the code for all the cuda functions used
 *
 */
 
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include "ParticleSystem.h"
#include "Util/SVector3.h"
#include "Util/SSphere.h"
#include "BVHNode.h"

#define THREADS_PER_BLOCK 32

// Variables needed for moveParticle(): ParticleSystem.particles[], ParticleSystem.speed,
   // particle index, float "time" variable, GRAVITY constant
// Variables needed for resetParticle(): ParticleSystem.particles[], particle index,
   // SVector3 translation, rbFloat() & rFloat(), SPEED & YSPEED constants
   
typedef struct {
   Particle particles[MAX_PARTICLES];
   int numParticles;
   float speed;
   float random;
   SVector3 Translation;
} CudaParticleSystem;


__device__ float rFloat(hiprandState *randStates)  {
   int id = (blockIdx.x * blockDim.x) + threadIdx.x;
   hiprandState localState = randStates[id];
   float randNum = (float)hiprand_uniform(&localState);
   randStates[id] = localState;
   
   return randNum;
}


__device__ float rbFloat(hiprandState *randStates) {
   return rFloat(randStates) * 2 - 1;
}


__device__ void moveParticle(Particle *particle, float speed, float time) {
   particle->velocity.Y -= GRAVITY * time;

   particle->sphere.center.X += particle->velocity.X * time * speed; 
   particle->sphere.center.Y += particle->velocity.Y * time * speed; 
   particle->sphere.center.Z += particle->velocity.Z * time * speed; 
}


__device__ void resetParticle(hiprandState *randStates, Particle *particle, SVector3 Translation, float random, float time) {
   particle->sphere.center.X = Translation.X + rbFloat(randStates) * random;
   particle->sphere.center.Y = Translation.Y + rbFloat(randStates) * random;
   particle->sphere.center.Z = Translation.Z + rbFloat(randStates) * random;
   
   particle->velocity.X = rbFloat(randStates) * SPEED;
   particle->velocity.Y = -rFloat(randStates) * YSPEED;
   particle->velocity.Z = rbFloat(randStates) * SPEED;
}


__global__ void update(hiprandState *randStates, CudaParticleSystem *cpsys, float time) {
   int index = (blockIdx.x * blockDim.x) + threadIdx.x;
   Particle curParticle = cpsys->particles[index];
   
   moveParticle(&curParticle, cpsys->speed, time);
   
   if(curParticle.sphere.center.Y < -2) {
      hiprand_init(1234, index, 0, &randStates[index]);
      resetParticle(randStates, &curParticle, cpsys->Translation, cpsys->random, time);
   }
   
   cpsys->particles[index] = curParticle;
}


extern "C" void cudaUpdate(ParticleSystem *psys, float time) {
   int num_blocks = 0;
   hiprandState *randStates;

   CudaParticleSystem *cpsys_device;
   
   hipMalloc((void **)&cpsys_device, sizeof(CudaParticleSystem));
   hipMalloc((void **)&randStates, psys->numParticles * sizeof(hiprandState));
   
   hipMemcpy(cpsys_device->particles, psys->particles, sizeof(Particle) * psys->numParticles, hipMemcpyHostToDevice);
   hipMemcpy(&cpsys_device->numParticles, &psys->numParticles, sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(&cpsys_device->speed, &psys->speed, sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(&cpsys_device->random, &psys->random, sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(&cpsys_device->Translation, &psys->Translation, sizeof(SVector3), hipMemcpyHostToDevice);
   
   num_blocks = psys->numParticles / THREADS_PER_BLOCK + 1;
   
   update<<<THREADS_PER_BLOCK, num_blocks>>>(randStates, cpsys_device, time);

   hipMemcpy(psys->particles, cpsys_device->particles, sizeof(Particle) * psys->numParticles, hipMemcpyDeviceToHost);
   
   hipFree(cpsys_device);
   hipFree(randStates);
}



__device__ int checkTriangle(SVector3 A, SVector3 B, SVector3 C, SVector3 center, float radius, SVector3 vel)
{
  SVector3 normal = ((A - C).crossProduct(A - B));
  normal /= normal.length();

  SVector3 dirA = (A - (center - (normal * -radius)));
  SVector3 dirB = (A - (center + (normal * -radius)));

  float dot1 = dirA.dotProduct(normal);
  float dot2 = dirB.dotProduct(normal);

  if (dot1 > 0 && dot2 > 0 || dot1 < 0 && dot2 < 0)
    return 0;
  return 1;
}


__device__ SSphere* checkHit(CUDA_BVH* bvh, SSphere sphere)
{
   int queue[CUDABVHSIZE];
   queue[0] = 0; // add the head of the bvh to the queue
   int start = 0, end = 1;

   while (start != end) // while the queue isn't empty
   {
      SSphere tocheck = bvh[queue[start]].hsphere;
      if (tocheck.collidesWith(sphere))
      { 
         if (bvh[queue[start]].lIndex == -1 && bvh[queue[start]].rIndex == -1) // we found a leaf
            return &bvh[queue[start]].hsphere;
         // else
         queue[end++] = bvh[queue[start]].lIndex;
         queue[end++] = bvh[queue[start]].rIndex;
      }

      start++;
   }
   return NULL;
}

__global__ void collideWithBVH_kernel(CudaParticleSystem *cpsys, int num_p, CUDA_BVH* bvh, float bounce, float size)
{
  Particle part = cpsys->particles[blockIdx.x * blockDim.x + threadIdx.x];
  SSphere* hit = checkHit(bvh, part.sphere);

  if (hit != NULL && !hit->isEmpty() && checkTriangle(hit->A, hit->B, hit->C, part.sphere.center, part.sphere.radius, part.velocity))
  {
    float len = part.velocity.length();
    SVector3 dir = (part.sphere.center) - hit->center; 
    dir /= dir.length();
    dir *= len * bounce;        

    part.velocity = dir;
    part.sphere.center += (part.velocity) * size;
  }
  
  cpsys->particles[blockIdx.x * blockDim.x + threadIdx.x] = part;
}

extern "C" void CUDAcollideWithBVH(ParticleSystem *psys, CUDA_BVH* bvh)
{
   // step 1: copy the particles into a CUDA-compatible format
   CudaParticleSystem *cpsys_device;
   CUDA_BVH *cuda_bvh;
   
   hipMalloc((void **)&cpsys_device, sizeof(CudaParticleSystem));
   hipMalloc((void **)&cuda_bvh, CUDABVHSIZE * sizeof(CUDA_BVH));
   
   hipMemcpy(cuda_bvh, bvh, CUDABVHSIZE * sizeof(CUDA_BVH), hipMemcpyHostToDevice);
   hipMemcpy(cpsys_device->particles, psys->particles, sizeof(Particle) * psys->numParticles, hipMemcpyHostToDevice);
   hipMemcpy(&cpsys_device->numParticles, &psys->numParticles, sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(&cpsys_device->speed, &psys->speed, sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(&cpsys_device->random, &psys->random, sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(&cpsys_device->Translation, &psys->Translation, sizeof(SVector3), hipMemcpyHostToDevice);
 
   // step 2: call the kernel
   int num_blocks = psys->numParticles / THREADS_PER_BLOCK + 1;
   collideWithBVH_kernel<<<THREADS_PER_BLOCK, num_blocks>>>(cpsys_device, psys->numParticles, cuda_bvh, psys->bounce, psys->size);
   hipMemcpy(psys->particles, cpsys_device->particles, sizeof(Particle) * psys->numParticles, hipMemcpyDeviceToHost);
   
   hipFree(cpsys_device);
   hipFree(cuda_bvh);
}

__global__ void collideWith_kernel(CudaParticleSystem *cpsys, SSphere* spheres, int numspheres, float bounce, float size) {
  Particle part = cpsys->particles[blockIdx.x * blockDim.x + threadIdx.x];

  for (int j = 0; j < numspheres; j++)
  {
    if (spheres[j].collidesWith(part.sphere) && 
        checkTriangle(spheres[j].A, spheres[j].B, spheres[j].C, part.sphere.center, part.sphere.radius, part.velocity))
    {
      float len = part.velocity.length();
      SVector3 dir = (part.sphere.center) - spheres[j].center; 
      dir /= dir.length();
      dir *= len * bounce;        

      part.velocity = dir;
      part.sphere.center += (part.velocity) * size;

      break;
    }
  }

  cpsys->particles[blockIdx.x * blockDim.x + threadIdx.x] = part;
}

extern "C" void CUDAcollideWith(ParticleSystem *psys, std::vector<SSphere> spheres) {
   CudaParticleSystem *cpsys_device;
   SSphere *cu_spheres;
   
   hipMalloc((void **)&cpsys_device, sizeof(CudaParticleSystem));
   hipMalloc((void **)&cu_spheres, sizeof(SSphere) * spheres.size());
   
   hipMemcpy(cpsys_device->particles, psys->particles, sizeof(Particle) * psys->numParticles, hipMemcpyHostToDevice);
   hipMemcpy(&cpsys_device->numParticles, &psys->numParticles, sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(&cpsys_device->speed, &psys->speed, sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(&cpsys_device->random, &psys->random, sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(&cpsys_device->Translation, &psys->Translation, sizeof(SVector3), hipMemcpyHostToDevice);
   hipMemcpy(cu_spheres, &spheres[0], sizeof(SSphere) * spheres.size(), hipMemcpyHostToDevice);
   
 
   int num_blocks = psys->numParticles / THREADS_PER_BLOCK + 1;
   collideWith_kernel<<<THREADS_PER_BLOCK, num_blocks>>>(cpsys_device, cu_spheres, spheres.size(), psys->bounce, psys->size);
   hipMemcpy(psys->particles, cpsys_device->particles, sizeof(Particle) * psys->numParticles, hipMemcpyDeviceToHost);

   hipFree(cpsys_device);
   hipFree(cu_spheres);
}
